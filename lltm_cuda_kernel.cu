#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
  return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
  const auto s = sigmoid(z);
  return (1.0 - s) * s;
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_tanh(scalar_t z) {
  const auto t = tanh(z);
  return 1 - (t * t);
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t elu(scalar_t z, scalar_t alpha = 1.0) {
  return fmax(0.0, z) + fmin(0.0, alpha * (exp(z) - 1.0));
}

template <typename scalar_t>
__device__ __forceinline__ scalar_t d_elu(scalar_t z, scalar_t alpha = 1.0) {
  const auto e = exp(z);
  const auto d_relu = z < 0.0 ? 0.0 : 1.0;
  return d_relu + (((alpha * (e - 1.0)) < 0.0) ? (alpha * e) : 0.0);
}

std::vector<torch::Tensor> lltm_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor bias,
    torch::Tensor old_h,
    torch::Tensor old_cell) {
  auto X = torch::cat({old_h, input}, /*dim=*/1);
  auto gates = torch::addmm(bias, X, weights.transpose(0, 1));

  const auto batch_size = old_cell.size(0);
  const auto state_size = old_cell.size(1);

  auto new_h = torch::zeros_like(old_cell);
  auto new_cell = torch::zeros_like(old_cell);
  auto input_gate = torch::zeros_like(old_cell);
  auto output_gate = torch::zeros_like(old_cell);
  auto candidate_cell = torch::zeros_like(old_cell);

  const int threads = 1024;
  const dim3 blocks((state_size + threads - 1) / threads, batch_size);

  AT_DISPATCH_FLOATING_TYPES(gates.type(), "lltm_forward_cuda", ([&] {
    lltm_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        gates.data<scalar_t>(),
        old_cell.data<scalar_t>(),
        new_h.data<scalar_t>(),
        new_cell.data<scalar_t>(),
        input_gate.data<scalar_t>(),
        output_gate.data<scalar_t>(),
        candidate_cell.data<scalar_t>(),
        state_size);
  }));

  return {new_h, new_cell, input_gate, output_gate, candidate_cell, X, gates};
}

template <typename scalar_t>
__global__ void lltm_cuda_forward_kernel(
    const scalar_t* __restrict__ gates,
    const scalar_t* __restrict__ old_cell,
    scalar_t* __restrict__ new_h,
    scalar_t* __restrict__ new_cell,
    scalar_t* __restrict__ input_gate,
    scalar_t* __restrict__ output_gate,
    scalar_t* __restrict__ candidate_cell,
    size_t state_size) {
  const int column = blockIdx.x * blockDim.x + threadIdx.x;
  const int index = blockIdx.y * state_size + column;
  const int gates_row = blockIdx.y * (state_size * 3);
  if (column < state_size) {
    input_gate[index] = sigmoid(gates[gates_row + column]);
    output_gate[index] = sigmoid(gates[gates_row + state_size + column]);
    candidate_cell[index] = elu(gates[gates_row + 2 * state_size + column]);
    new_cell[index] =
        old_cell[index] + candidate_cell[index] * input_gate[index];
    new_h[index] = tanh(new_cell[index]) * output_gate[index];
  }
}

